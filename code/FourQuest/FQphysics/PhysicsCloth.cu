#include "hip/hip_runtime.h"
#include "PhysicsCloth.h"

#include "EngineDataConverter.h"

__device__ physx::PxVec2 Sub(const physx::PxVec2& lhs, const physx::PxVec2& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y };
}

__device__ physx::PxVec4 Sub(const physx::PxVec4& lhs, const physx::PxVec4& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w };
}

__device__ physx::PxVec3 cross(const physx::PxVec4& a, const physx::PxVec4& b) {
	return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x };
}

__device__ float DotProduct(const physx::PxVec3& a, const physx::PxVec3& b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ void NormalizeVector(physx::PxVec3& vec) {
	float length = sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
	if (length > 0) {
		vec.x /= length;
		vec.y /= length;
		vec.z /= length;
	}
}

#pragma region UpdateID3D11VertexBuffer
// CUDA Ŀ�� �Լ� ����
__global__ void UpdateVertex(
	physx::PxVec4* vertices,
	physx::PxVec2* uvs,
	unsigned int vertexSize,
	unsigned int* indices,
	unsigned int indexSize,
	fq::physics::Vertex* buffer)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= indexSize / 3) return;

	unsigned int i0 = indices[idx * 3];
	unsigned int i1 = indices[idx * 3 + 1];
	unsigned int i2 = indices[idx * 3 + 2];

	physx::PxVec4 v0 = vertices[i0];
	physx::PxVec4 v1 = vertices[i1];
	physx::PxVec4 v2 = vertices[i2];
	v0.z = -v0.z;
	v1.z = -v1.z;
	v2.z = -v2.z;

	physx::PxVec2 uv0 = uvs[i0];
	physx::PxVec2 uv1 = uvs[i1];
	physx::PxVec2 uv2 = uvs[i2];

	physx::PxVec4 edge1 = Sub(v1, v0);
	physx::PxVec4 edge2 = Sub(v2, v0);

	physx::PxVec2 deltaUV1 = Sub(uv1, uv0);
	physx::PxVec2 deltaUV2 = Sub(uv2, uv0);

	float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

	physx::PxVec3 tangent;
	tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
	tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
	tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
	NormalizeVector(tangent);

	physx::PxVec3 bitangent;
	bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
	bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
	bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
	NormalizeVector(bitangent);

	physx::PxVec3 normal = cross(edge1, edge2);
	NormalizeVector(normal);

	buffer[i0].Pos.x = v0.x;
	buffer[i0].Pos.y = v0.y;
	buffer[i0].Pos.z = v0.z;
	buffer[i1].Pos.x = v1.x;
	buffer[i1].Pos.y = v1.y;
	buffer[i1].Pos.z = v1.z;
	buffer[i2].Pos.x = v2.x;
	buffer[i2].Pos.y = v2.y;
	buffer[i2].Pos.z = v2.z;

	buffer[i0].Normal.x = normal.x;
	buffer[i0].Normal.y = normal.y;
	buffer[i0].Normal.z = normal.z;
	buffer[i1].Normal.x = normal.x;
	buffer[i1].Normal.y = normal.y;
	buffer[i1].Normal.z = normal.z;
	buffer[i2].Normal.x = normal.x;
	buffer[i2].Normal.y = normal.y;
	buffer[i2].Normal.z = normal.z;

	buffer[i0].Tangent.x = tangent.x;
	buffer[i0].Tangent.y = tangent.y;
	buffer[i0].Tangent.z = tangent.z;
	buffer[i1].Tangent.x = tangent.x;
	buffer[i1].Tangent.y = tangent.y;
	buffer[i1].Tangent.z = tangent.z;
	buffer[i2].Tangent.x = tangent.x;
	buffer[i2].Tangent.y = tangent.y;
	buffer[i2].Tangent.z = tangent.z;
}
#pragma endregion

#pragma region SetID3D11BufferVertexNormal
__global__ void processVerticesKernel(unsigned int* sameVerticesFirst, unsigned int* sameVerticesSecond,
	fq::physics::Vertex* buffer, int size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {
		int first = sameVerticesFirst[idx];
		int second = sameVerticesSecond[idx];

		// �߰����� ����Ͽ� ���� ������ �и�
		float3 normalTemp, tangentTemp, biTangentTemp;

		normalTemp.x = buffer[first].Normal.x + buffer[second].Normal.x;
		normalTemp.y = buffer[first].Normal.y + buffer[second].Normal.y;
		normalTemp.z = buffer[first].Normal.z + buffer[second].Normal.z;

		tangentTemp.x = buffer[first].Tangent.x + buffer[second].Tangent.x;
		tangentTemp.y = buffer[first].Tangent.y + buffer[second].Tangent.y;
		tangentTemp.z = buffer[first].Tangent.z + buffer[second].Tangent.z;

		buffer[first].Normal.x = normalTemp.x;
		buffer[first].Normal.y = normalTemp.y;
		buffer[first].Normal.z = normalTemp.z;
		buffer[second].Normal.x = normalTemp.x;
		buffer[second].Normal.y = normalTemp.y;
		buffer[second].Normal.z = normalTemp.z;

		buffer[first].Tangent.x = tangentTemp.x;
		buffer[first].Tangent.y = tangentTemp.y;
		buffer[first].Tangent.z = tangentTemp.z;
		buffer[second].Tangent.x = tangentTemp.x;
		buffer[second].Tangent.y = tangentTemp.y;
		buffer[second].Tangent.z = tangentTemp.z;
	}
}
#pragma endregion

#pragma region SetClothParticleWorldTransform
__device__ DirectX::SimpleMath::Vector3 multiply(DirectX::SimpleMath::Matrix& mat, const DirectX::SimpleMath::Vector3& vec)
{
	DirectX::SimpleMath::Vector3 result;
	result.x = mat.m[0][0] * vec.x + mat.m[1][0] * vec.y + mat.m[2][0] * vec.z + mat.m[3][0] * 1.0f;
	result.y = mat.m[0][1] * vec.x + mat.m[1][1] * vec.y + mat.m[2][1] * vec.z + mat.m[3][1] * 1.0f;
	result.z = mat.m[0][2] * vec.x + mat.m[1][2] * vec.y + mat.m[2][2] * vec.z + mat.m[3][2] * 1.0f;
	return result;
}

// Ŀ�� �Լ�
__global__ void TransformVertices(
	physx::PxVec4* particle,
	DirectX::SimpleMath::Matrix previousTransformInverse,
	DirectX::SimpleMath::Matrix newTransform,
	int vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexCount) return;

	DirectX::SimpleMath::Vector3 vertex;
	vertex.x = particle[idx].x;
	vertex.y = particle[idx].y;
	vertex.z = -particle[idx].z;

	// ���� worldTransform�� ����� ����
	vertex = multiply(previousTransformInverse, vertex);

	// ���ο� worldTransform ����
	vertex = multiply(newTransform, vertex);

	// ��ȯ�� vertex ����
	particle[idx].x = vertex.x;
	particle[idx].y = vertex.y;
	particle[idx].z = -vertex.z;
}
#pragma endregion


void MulMatrixVector3(const DirectX::SimpleMath::Matrix& mat, DirectX::SimpleMath::Vector3& vec)
{
	DirectX::SimpleMath::Vector3 result;
	result.x = mat._11 * vec.x + mat._21 * vec.y + mat._31 * vec.z + mat._41;
	result.y = mat._12 * vec.x + mat._22 * vec.y + mat._32 * vec.z + mat._42;
	result.z = mat._13 * vec.x + mat._23 * vec.y + mat._33 * vec.z + mat._43;
	vec = result;
}

namespace fq::physics
{
	PhysicsCloth::PhysicsCloth(unsigned int id, unsigned int layerNumber)
		: mID()
		, mLayNumber()
		, mWorldTransform{}
		, mParticleSystem(nullptr)
		, mClothBuffer(nullptr)
		, mPBDMaterial(nullptr)
	{
	}

	PhysicsCloth::~PhysicsCloth()
	{
	}

	bool PhysicsCloth::Initialize(const PhysicsClothInfo& info, physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		int deviceCount;
		hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
		if (cudaStatus != hipSuccess || deviceCount == 0) {
			std::cerr << "CUDA ����̽� �ʱ�ȭ ����" << std::endl;
			return false;
		}

		cudaStatus = hipSetDevice(0); // ù ��° CUDA ����̽� ���
		if (cudaStatus != hipSuccess) {
			std::cerr << "CUDA ����̽� ���� ����" << std::endl;
			return false;
		}

		if (cudaContextManager == nullptr)
			return false;

		settingInfoData(info);
		extractSpringsData();

		// ���(Material) ����
		mPBDMaterial = physics->createPBDMaterial(
			info.materialInfo.friction,
			info.materialInfo.damping,
			info.materialInfo.adhesion,
			info.materialInfo.viscosity,
			info.materialInfo.vorticityConfinement,
			info.materialInfo.surfaceTension,
			info.materialInfo.cohesion,
			info.materialInfo.lift,
			info.materialInfo.drag,
			info.materialInfo.cflCoefficient,
			info.materialInfo.gravityScale);

		createClothParticle(physics, scene, cudaContextManager);

		return true;
	}

	bool PhysicsCloth::UpdatePhysicsCloth(physx::PxCudaContextManager* cudaContextManager)
	{
		physx::PxVec4* paticle = mClothBuffer->getPositionInvMasses();

		if (!updatePhysXDataToID3DBuffer(mVertices, mIndices, mUV, mCudaVertexResource, paticle)) return false;
		if (!updateNormalToID3DBuffer(mSameVertices, mVertices.size(), mCudaVertexResource)) return false;

		return true;
	}

	void PhysicsCloth::settingInfoData(const PhysicsClothInfo& info)
	{
		RegisterD3D11VertexBufferWithCUDA((ID3D11Buffer*)info.vertexBuffer);
		RegisterD3D11IndexBufferWithCUDA((ID3D11Buffer*)info.indexBuffer);

		mWorldTransform = info.worldTransform;
		mTotalClothMass = info.totalClothMass;

		mIndices.resize(info.indexSize);
		memcpy(mIndices.data(), info.indices, info.indexSize * sizeof(unsigned int));

		mUV.resize(info.vertexSize);
		mVertices.resize(info.vertexSize);

		for (int i = 0; i < info.vertexSize; i++)
		{
			mVertices[i].x = info.vertices[i].x;
			mVertices[i].y = info.vertices[i].y;
			mVertices[i].z = info.vertices[i].z;
			mUV[i] = info.uvs[i];
		}

		for (auto& vertex : mVertices)
		{
			MulMatrixVector3(mWorldTransform, vertex);
		}

		for (int i = 0; i < info.vertexSize; i++)
		{
			mVertices[i].z = -mVertices[i].z;
		}
	}

	bool areVerticesEqual(const DirectX::SimpleMath::Vector3& vertex1, const DirectX::SimpleMath::Vector3& vertex2, float epsilon = 1e-6) {
		return (std::abs(vertex1.x - vertex2.x) < epsilon) &&
			(std::abs(vertex1.y - vertex2.y) < epsilon) &&
			(std::abs(vertex1.z - vertex2.z) < epsilon);
	}

	void PhysicsCloth::extractSpringsData()
	{
		// �ﰢ�� ������ �ε����� ��ȸ
		for (size_t i = 0; i < mIndices.size(); i += 3)
		{
			unsigned int v1 = mIndices[i];
			unsigned int v2 = mIndices[i + 1];
			unsigned int v3 = mIndices[i + 2];

			// ���ĵ� ������ vertex ���� �߰��Ͽ� �ߺ��� ���� ����
			auto addEdge = [this](unsigned int a, unsigned int b)
				{
					if (a > b) std::swap(a, b);
					mSprings.insert({ a, b });
				};

			addEdge(v1, v2);
			addEdge(v2, v3);
			addEdge(v3, v1);
		}

		// ���� ��ġ�� ���ؽ��� ���� ����
		mSameVertices.reserve(mVertices.size() / 3);
		for (int i = 0; i < mVertices.size(); i++)
		{
			for (int j = i + 1; j < mVertices.size(); j++)
			{
				if (areVerticesEqual(mVertices[i], mVertices[j]))
				{
					mSprings.insert({ i, j });
					mSameVertices.push_back({ i, j });
				}
			}
		}
	}

	void PhysicsCloth::createClothParticle(physx::PxPhysics* physics, physx::PxScene* scene, physx::PxCudaContextManager* cudaContextManager)
	{
		// ���� �� ������, �ﰢ���� ���� ���
		const physx::PxU32 numParticles = mVertices.size();	// ���� ����
		const physx::PxU32 numSprings = mSprings.size();	// ���� �ϳ��� �̿��ϴ� ���ڵ鿡 ������ ���� �����µ�, �� ������ ����
		const physx::PxU32 numTriangles = mIndices.size() / 3;	// �ﰢ�� ����

		// ���� �ý����� ����
		const physx::PxReal particleMass = mTotalClothMass / mVertices.size();
		const physx::PxReal restOffset = 2.f;

		// ���� �ý��� ����
		mParticleSystem = physics->createPBDParticleSystem(*cudaContextManager);

		mParticleSystem->setRestOffset(1.f);
		mParticleSystem->setContactOffset(restOffset + 0.02f);
		mParticleSystem->setParticleContactOffset(restOffset + 0.02f);
		mParticleSystem->setSolidRestOffset(restOffset);

		// ���� ���� �ý��� �߰�
		scene->addActor(*mParticleSystem);

		// ������ ���¸� �����ϴ� ���� ����
		const physx::PxU32 particlePhase = mParticleSystem->createPhase(mPBDMaterial, physx::PxParticlePhaseFlags(
			physx::PxParticlePhaseFlag::eParticlePhaseSelfCollideFilter | physx::PxParticlePhaseFlag::eParticlePhaseSelfCollide));

		mClothBufferHelper = physx::ExtGpu::PxCreateParticleClothBufferHelper(1, numTriangles, numSprings, numParticles, cudaContextManager);
		// ����, ������ �ﰢ���� ���¸� �����ϱ� ���� ���� �Ҵ�
		unsigned int* phase = cudaContextManager->allocPinnedHostBuffer<physx::PxU32>(numParticles);
		physx::PxVec4* positionInvMass = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);
		physx::PxVec4* velocity = cudaContextManager->allocPinnedHostBuffer<physx::PxVec4>(numParticles);

		// cloth�� ������ ��ƼŬ�� ������ �����͸� ������� �ϴ� Cloth Particle Buffer ����
		settingParticleBuffer(numSprings, numTriangles, numParticles, particlePhase, particleMass, phase, positionInvMass, velocity);

		// cloth ����
		createCloth(numParticles, cudaContextManager, phase, positionInvMass, velocity);
	}

	float calculateVectorMagnitude(const DirectX::SimpleMath::Vector3& point1, const DirectX::SimpleMath::Vector3& point2) {
		float dx = point2.x - point1.x;
		float dy = point2.y - point1.y;
		float dz = point2.z - point1.z;

		return std::sqrt(dx * dx + dy * dy + dz * dz);
	}

	void PhysicsCloth::settingParticleBuffer(
		const physx::PxU32& numSprings,
		const physx::PxU32& numTriangles,
		const physx::PxU32& numParticles,
		const physx::PxU32& particlePhase,
		const physx::PxReal& particleMass,
		unsigned int* phase,
		physx::PxVec4* positionInvMass,
		physx::PxVec4* velocity)
	{
		const physx::PxReal stretchStiffness = 100.f;
		const physx::PxReal shearStiffness = 100.f;
		const physx::PxReal springDamping = 0.1f;

		// ��ƼŬ ������ �� Ʈ���̾ޱ� ����
		physx::PxArray<physx::PxParticleSpring> springs;
		springs.reserve(numSprings);
		physx::PxArray<physx::PxU32> triangles;
		triangles.reserve(numTriangles * 3);

		// ���� ���� ����
		for (int i = 0; i < numParticles; i++)
		{
			positionInvMass[i] = physx::PxVec4(mVertices[i].x, mVertices[i].y + 300.f, mVertices[i].z, 1.f / particleMass);
			phase[i] = particlePhase;
			velocity[i] = physx::PxVec4(0.f);
		}

		// ������ �߰�
		for (auto line : mSprings)
		{
			physx::PxParticleSpring spring = { line.first, line.second, calculateVectorMagnitude(mVertices[line.first], mVertices[line.second]), stretchStiffness, springDamping, 0 };
			springs.pushBack(spring);
		}

		// �ﰢ�� �߰�
		for (int i = 0; i < mIndices.size(); i += 3)
		{
			triangles.pushBack(mIndices[i]);
			triangles.pushBack(mIndices[i + 1]);
			triangles.pushBack(mIndices[i + 2]);
		}

		// ������ ������ �� �ﰢ�� ���� ������ �����Ǿ����� Ȯ��
		PX_ASSERT(numSprings == springs.size());
		PX_ASSERT(numTriangles == triangles.size() / 3);

		// õ���� ���ۿ� ������ �߰�
		mClothBufferHelper->addCloth(0.f, 0.f, 0.f, triangles.begin(), numTriangles, springs.begin(), numSprings, positionInvMass, numParticles);
	}

	void PhysicsCloth::createCloth(
		const physx::PxU32& numParticles,
		physx::PxCudaContextManager* cudaContextManager,
		unsigned int* phase,
		physx::PxVec4* positionInvMass,
		physx::PxVec4* velocity)
	{
		// ������ ���¸� ��Ÿ���� ���� ����
		physx::ExtGpu::PxParticleBufferDesc bufferDesc;
		bufferDesc.maxParticles = numParticles;
		bufferDesc.numActiveParticles = numParticles;
		bufferDesc.positions = positionInvMass;
		bufferDesc.velocities = velocity;
		bufferDesc.phases = phase;

		// õ���� ���� ��������
		const physx::PxParticleClothDesc& clothDesc = mClothBufferHelper->getParticleClothDesc();

		// ���� õ���� ��ó���� ����
		physx::PxParticleClothPreProcessor* clothPreProcessor = PxCreateParticleClothPreProcessor(cudaContextManager);

		// ���� õ�� ���� �� ó��
		physx::PxPartitionedParticleCloth output;
		clothPreProcessor->partitionSprings(clothDesc, output);
		clothPreProcessor->release();

		// õ�� ���� ����
		mClothBuffer = physx::ExtGpu::PxCreateAndPopulateParticleClothBuffer(bufferDesc, clothDesc, output, cudaContextManager);
		mParticleSystem->addParticleBuffer(mClothBuffer);

		// ���� ����
		mClothBufferHelper->release();

		// �Ҵ�� �޸� ����
		cudaContextManager->freePinnedHostBuffer(positionInvMass);
		cudaContextManager->freePinnedHostBuffer(velocity);
		cudaContextManager->freePinnedHostBuffer(phase);
	}

	void PhysicsCloth::GetPhysicsCloth(PhysicsClothGetData& data)
	{
		data.worldTransform = mWorldTransform;
	}

	physx::PxVec4 multiply(const physx::PxMat44& mat, const physx::PxVec4& vec)  // 4x4 ��İ� PxVec4�� ���ϴ� �Լ�
	{
		physx::PxVec4 result;
		result.x = mat(0, 0) * vec.x + mat(0, 1) * vec.y + mat(0, 2) * vec.z + mat(0, 3) * vec.w;
		result.y = mat(1, 0) * vec.x + mat(1, 1) * vec.y + mat(1, 2) * vec.z + mat(1, 3) * vec.w;
		result.z = mat(2, 0) * vec.x + mat(2, 1) * vec.y + mat(2, 2) * vec.z + mat(2, 3) * vec.w;
		result.w = mat(3, 0) * vec.x + mat(3, 1) * vec.y + mat(3, 2) * vec.z + mat(3, 3) * vec.w;
		return result;
	}

	bool PhysicsCloth::SetPhysicsCloth(const PhysicsClothSetData& data)
	{
		physx::PxVec4* paticle = mClothBuffer->getPositionInvMasses();

		if (!updateWorldTransformToID3DBuffer(mWorldTransform, data.worldTransform, mVertices.size(), paticle)) return false;

		return true;
	}

	bool PhysicsCloth::RegisterD3D11VertexBufferWithCUDA(ID3D11Buffer* buffer)
	{
		hipError_t cudaStatus = cudaGraphicsD3D11RegisterResource(&mCudaVertexResource, buffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "Direct3D ���ҽ� ��� ����" << std::endl;
			return false;
		}
		return true;
	}

	bool PhysicsCloth::RegisterD3D11IndexBufferWithCUDA(ID3D11Buffer* buffer)
	{
		hipError_t cudaStatus = cudaGraphicsD3D11RegisterResource(&mCudaIndexResource, buffer, hipGraphicsRegisterFlagsNone);
		if (cudaStatus != hipSuccess)
		{
			std::cerr << "Direct3D ���ҽ� ��� ����" << std::endl;
			return false;
		}
		return true;
	}

#pragma region CudaFunction
	bool PhysicsCloth::updatePhysXDataToID3DBuffer(
		std::vector<DirectX::SimpleMath::Vector3>& vertices,
		std::vector<unsigned int>& indices,
		std::vector<DirectX::SimpleMath::Vector2> uvs,
		hipGraphicsResource* directXVertexBuffer,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (indices.size() / 3 + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &directXVertexBuffer);
		if (cudaStatus != hipSuccess) {
			std::cerr << "hipGraphicsMapResources ����: " << hipGetErrorString(cudaStatus) << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		hipGraphicsResourceGetMappedPointer(&devPtr, &size, directXVertexBuffer);
		if (cudaStatus != hipSuccess) {
			std::cerr << "hipGraphicsResourceGetMappedPointer ����: " << hipGetErrorString(cudaStatus) << std::endl;
			return false;
		}

		// GPU Memory�� �Ҵ��� ����
		physx::PxVec2* d_uvs = nullptr;
		unsigned int* d_indices = nullptr;

		// GPU Memory�� �Ҵ� �� ������ ����
		hipMalloc(&d_uvs, uvs.size() * sizeof(DirectX::SimpleMath::Vector2));
		hipMalloc(&d_indices, indices.size() * sizeof(unsigned int));
		hipMemcpy(d_uvs, uvs.data(), uvs.size() * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		// CUDA �Լ� ����
		UpdateVertex << <blocksPerGrid, threadsPerBlock >> > (
			particle, d_uvs, deviceVertexSize, d_indices, deviceIndexSize, (PhysicsVertex*)devPtr);
		hipDeviceSynchronize();

		// CUDA ���ҽ��� �����
		hipGraphicsUnmapResources(1, &directXVertexBuffer);

		// �޸� ����
		hipFree(d_uvs);
		hipFree(d_indices);

		return true;
	}

	bool PhysicsCloth::updateNormalToID3DBuffer(
		std::vector<std::pair<unsigned int, unsigned int>>& sameVertices,
		unsigned int vertexSize,
		hipGraphicsResource* directXVertexBuffer)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &directXVertexBuffer);
		if (cudaStatus != hipSuccess) {
			std::cerr << "hipGraphicsMapResources ����: " << hipGetErrorString(cudaStatus) << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		hipGraphicsResourceGetMappedPointer(&devPtr, &size, directXVertexBuffer);
		if (cudaStatus != hipSuccess) {
			std::cerr << "hipGraphicsResourceGetMappedPointer ����: " << hipGetErrorString(cudaStatus) << std::endl;
			return false;
		}

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize / 3 + threadsPerBlock - 1) / threadsPerBlock;

		blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

		std::vector<unsigned int> firstVertex;
		std::vector<unsigned int> secondVertex;
		firstVertex.resize(sameVertices.size());
		secondVertex.resize(sameVertices.size());
		for (int i = 0; i < sameVertices.size(); i++)
		{
			firstVertex.push_back(sameVertices[i].first);
			secondVertex.push_back(sameVertices[i].second);
		}

		unsigned int* d_firstVertex;
		unsigned int* d_secondVertex;

		hipMalloc(&d_firstVertex, firstVertex.size() * sizeof(unsigned int));
		hipMalloc(&d_secondVertex, secondVertex.size() * sizeof(unsigned int));
		hipMemcpy(d_firstVertex, firstVertex.data(), firstVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_secondVertex, secondVertex.data(), secondVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		processVerticesKernel << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (PhysicsVertex*)devPtr, mVertices.size());
		hipDeviceSynchronize();

		// CUDA ���ҽ��� �����
		hipGraphicsUnmapResources(1, &directXVertexBuffer);

		// �޸� ����
		hipFree(d_firstVertex);
		hipFree(d_secondVertex);

		return true;
	}

	bool PhysicsCloth::updateWorldTransformToID3DBuffer(
		const DirectX::SimpleMath::Matrix& prevWorldTransform,
		const DirectX::SimpleMath::Matrix& nextWorldTrnasform,
		const unsigned int vertexSize,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize / 3 + threadsPerBlock - 1) / threadsPerBlock;

		DirectX::SimpleMath::Matrix prevTransform = prevWorldTransform.Invert();
		DirectX::SimpleMath::Matrix nextTransform = nextWorldTrnasform;

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		TransformVertices << <blocksPerGrid, threadsPerBlock >> > (particle, prevMatrix, nextMatrix, vertexSize);
		hipDeviceSynchronize();

		return true;
	}
#pragma endregion

}