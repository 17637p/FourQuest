#include "hip/hip_runtime.h"
#include "CudaClothTool.h"

#include <hip/hip_runtime.h>
#include <cuda_d3d11_interop.h>
#include <>

#include <iostream>

#include "../FQCommon/FQCommonGraphics.h"

__device__ physx::PxVec2 Sub(const physx::PxVec2& lhs, const physx::PxVec2& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y };
}

__device__ physx::PxVec4 Sub(const physx::PxVec4& lhs, const physx::PxVec4& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w };
}

__device__ physx::PxVec3 cross(const physx::PxVec4& a, const physx::PxVec4& b) {
	return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x };
}

__device__ float DotProduct(const physx::PxVec3& a, const physx::PxVec3& b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ void NormalizeVector(physx::PxVec3& vec) {
	float length = sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
	if (length > 0) {
		vec.x /= length;
		vec.y /= length;
		vec.z /= length;
	}
}

#pragma region UpdateID3D11VertexBuffer
// CUDA Ŀ�� �Լ� ����
__global__ void UpdateVertex(
	physx::PxVec4* vertices,
	physx::PxVec2* uvs,
	unsigned int vertexSize,
	unsigned int* indices,
	unsigned int indexSize,
	Vertex* buffer)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= indexSize / 3) return;

	unsigned int i0 = indices[idx * 3];
	unsigned int i1 = indices[idx * 3 + 1];
	unsigned int i2 = indices[idx * 3 + 2];

	physx::PxVec4 v0 = vertices[i0];
	physx::PxVec4 v1 = vertices[i1];
	physx::PxVec4 v2 = vertices[i2];

	physx::PxVec2 uv0 = uvs[i0];
	physx::PxVec2 uv1 = uvs[i1];
	physx::PxVec2 uv2 = uvs[i2];

	physx::PxVec4 edge1 = Sub(v1, v0);
	physx::PxVec4 edge2 = Sub(v2, v0);

	physx::PxVec2 deltaUV1 = Sub(uv1, uv0);
	physx::PxVec2 deltaUV2 = Sub(uv2, uv0);

	float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

	physx::PxVec3 tangent;
	tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
	tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
	tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
	NormalizeVector(tangent);

	physx::PxVec3 bitangent;
	bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
	bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
	bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
	NormalizeVector(bitangent);

	physx::PxVec3 normal = cross(edge1, edge2);
	NormalizeVector(normal);

	buffer[i0].Pos.x = v0.x;
	buffer[i0].Pos.y = v0.y;
	buffer[i0].Pos.z = v0.z;
	buffer[i1].Pos.x = v1.x;
	buffer[i1].Pos.y = v1.y;
	buffer[i1].Pos.z = v1.z;
	buffer[i2].Pos.x = v2.x;
	buffer[i2].Pos.y = v2.y;
	buffer[i2].Pos.z = v2.z;

	buffer[i0].Normal.x = normal.x;
	buffer[i0].Normal.y = normal.y;
	buffer[i0].Normal.z = normal.z;
	buffer[i1].Normal.x = normal.x;
	buffer[i1].Normal.y = normal.y;
	buffer[i1].Normal.z = normal.z;
	buffer[i2].Normal.x = normal.x;
	buffer[i2].Normal.y = normal.y;
	buffer[i2].Normal.z = normal.z;

	buffer[i0].Tangent.x = tangent.x;
	buffer[i0].Tangent.y = tangent.y;
	buffer[i0].Tangent.z = tangent.z;
	buffer[i1].Tangent.x = tangent.x;
	buffer[i1].Tangent.y = tangent.y;
	buffer[i1].Tangent.z = tangent.z;
	buffer[i2].Tangent.x = tangent.x;
	buffer[i2].Tangent.y = tangent.y;
	buffer[i2].Tangent.z = tangent.z;
}
#pragma endregion

#pragma region SetID3D11BufferVertexNormal
__global__ void processVerticesKernel(unsigned int* sameVerticesFirst, unsigned int* sameVerticesSecond, Vertex* buffer, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int first = sameVerticesFirst[idx];
	int second = sameVerticesSecond[idx];

	if (first < size && second < size)
	{
		// �߰����� ����Ͽ� ���� ������ �и�
		float3 normalTemp, tangentTemp, biTangentTemp;

		normalTemp.x = buffer[first].Normal.x + buffer[second].Normal.x;
		normalTemp.y = buffer[first].Normal.y + buffer[second].Normal.y;
		normalTemp.z = buffer[first].Normal.z + buffer[second].Normal.z;

		tangentTemp.x = buffer[first].Tangent.x + buffer[second].Tangent.x;
		tangentTemp.y = buffer[first].Tangent.y + buffer[second].Tangent.y;
		tangentTemp.z = buffer[first].Tangent.z + buffer[second].Tangent.z;

		buffer[first].Normal.x = normalTemp.x;
		buffer[first].Normal.y = normalTemp.y;
		buffer[first].Normal.z = normalTemp.z;
		buffer[second].Normal.x = normalTemp.x;
		buffer[second].Normal.y = normalTemp.y;
		buffer[second].Normal.z = normalTemp.z;

		buffer[first].Tangent.x = tangentTemp.x;
		buffer[first].Tangent.y = tangentTemp.y;
		buffer[first].Tangent.z = tangentTemp.z;
		buffer[second].Tangent.x = tangentTemp.x;
		buffer[second].Tangent.y = tangentTemp.y;
		buffer[second].Tangent.z = tangentTemp.z;
	}
}
#pragma endregion

#pragma region SetClothParticleWorldTransform
struct SimpleVector3 {
	float x, y, z;
};

struct SimpleMatrix {
	float m[4][4];
};

__device__ SimpleVector3 multiply(SimpleMatrix& mat, const SimpleVector3& vec)
{
	SimpleVector3 result;
	result.x = mat.m[0][0] * vec.x + mat.m[1][0] * vec.y + mat.m[2][0] * vec.z + mat.m[3][0] * 1.0f;
	result.y = mat.m[0][1] * vec.x + mat.m[1][1] * vec.y + mat.m[2][1] * vec.z + mat.m[3][1] * 1.0f;
	result.z = mat.m[0][2] * vec.x + mat.m[1][2] * vec.y + mat.m[2][2] * vec.z + mat.m[3][2] * 1.0f;
	return result;
}

// Ŀ�� �Լ�
__global__ void TransformVertices(
	physx::PxVec4* particle,
	SimpleMatrix previousTransformInverse,
	SimpleMatrix newTransform,
	int vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexCount) return;

	SimpleVector3 vertex;
	vertex.x = particle[idx].x;
	vertex.y = particle[idx].y;
	vertex.z = particle[idx].z;

	// ���� worldTransform�� ����� ����
	vertex = multiply(previousTransformInverse, vertex);

	// ���ο� worldTransform ����
	vertex = multiply(newTransform, vertex);

	// ��ȯ�� vertex ����
	particle[idx].x = vertex.x;
	particle[idx].y = vertex.y;
	particle[idx].z = vertex.z;
}
#pragma endregion

#pragma region CopyFromGPUToCPU
__global__ void CopyVertexDataToCPU(
	physx::PxVec3* vertices,
	physx::PxVec2* uvs,
	SimpleMatrix worldTransform,
	Vertex* ID3D11VertexBuffer,
	unsigned int vertexSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexSize) return;

	SimpleVector3 vertex;

	vertex.x = ID3D11VertexBuffer[idx].Pos.x;
	vertex.y = ID3D11VertexBuffer[idx].Pos.y;
	vertex.z = ID3D11VertexBuffer[idx].Pos.z;

	uvs[idx].x = ID3D11VertexBuffer[idx].Tex.x;
	uvs[idx].y = ID3D11VertexBuffer[idx].Tex.y;

	vertex = multiply(worldTransform, vertex);

	vertices[idx].x = vertex.x;
	vertices[idx].y = vertex.y;
	vertices[idx].z = vertex.z;
}
#pragma endregion


namespace fq::physics
{
	bool CudaClothTool::copyVertexFromGPUToCPU(
		std::vector<DirectX::SimpleMath::Vector3>& vertices,
		std::vector<DirectX::SimpleMath::Vector2>& uvs,
		DirectX::SimpleMath::Matrix& worldTransform,
		hipGraphicsResource* ID3D11VertexBuffer)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t vertexSize = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &vertexSize, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		// GPU �޸� �Ҵ�
		physx::PxVec3* d_vertices;
		physx::PxVec2* d_uvs;
		hipMalloc(&d_vertices, vertexSize * sizeof(DirectX::SimpleMath::Vector3));
		hipMalloc(&d_uvs, vertexSize * sizeof(DirectX::SimpleMath::Vector2));

		SimpleMatrix Trnasform;
		memcpy(&Trnasform, &worldTransform, sizeof(SimpleMatrix));

		// CUDA �Լ� ����
		CopyVertexDataToCPU << < blocksPerGrid, threadsPerBlock >> > (d_vertices, d_uvs, Trnasform, (Vertex*)devPtr, vertexSize);
		cudaStatus = hipDeviceSynchronize(); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : "<< cudaStatus << ")" << std::endl;
			return false;
		}

		// ������ ������ ���� CPU �޸𸮷� ����
		hipMemcpy(vertices.data(), d_vertices, vertexSize * sizeof(DirectX::SimpleMath::Vector3), hipMemcpyKind::hipMemcpyDeviceToHost);
		hipMemcpy(uvs.data(), d_uvs, vertexSize * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyDeviceToHost);

		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		hipFree(d_vertices);
		hipFree(d_uvs);

		return true;
	}

	bool CudaClothTool::copyIndexFromGPUToCPU(
		std::vector<unsigned int>& indices, 
		hipGraphicsResource* ID3D11IndexBuffer)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11IndexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t indexSize = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &indexSize, ID3D11IndexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		hipMemcpy(indices.data(), devPtr, indexSize * sizeof(unsigned int), hipMemcpyKind::hipMemcpyDeviceToHost);

		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11IndexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}

	bool CudaClothTool::UpdatePhysXDataToID3DBuffer(
		std::vector<DirectX::SimpleMath::Vector3>& vertices, 
		std::vector<unsigned int>& indices, 
		std::vector<DirectX::SimpleMath::Vector2> uvs, 
		hipGraphicsResource* ID3D11VertexBuffer,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (indices.size() / 3 + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		unsigned int vertexSize = vertices.size();
		unsigned int indexSize = indices.size();

		if (size < sizeof(Vertex) * vertexSize) 
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] Mapped size is smaller than expected!" << std::endl;
			return false;
		}

		// GPU Memory�� �Ҵ��� ����
		physx::PxVec2* d_uvs;
		unsigned int* d_indices;

		// GPU Memory�� �Ҵ� �� ������ ����
		hipMalloc(&d_uvs, uvs.size() * sizeof(DirectX::SimpleMath::Vector2));
		hipMalloc(&d_indices, indices.size() * sizeof(unsigned int));
		hipMemcpy(d_uvs, uvs.data(), uvs.size() * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		// CUDA �Լ� ����
		UpdateVertex << <blocksPerGrid, threadsPerBlock >> > (
			particle, d_uvs, vertexSize, d_indices, indexSize, (Vertex*)devPtr);
		cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// �޸� ����
		hipFree(d_uvs);
		hipFree(d_indices);

		return true;
	}

	bool CudaClothTool::UpdateNormalToID3DBuffer(
		std::vector<std::pair<unsigned int, unsigned int>>& sameVertices,
		unsigned int vertexSize,
		hipGraphicsResource* ID3D11VertexBuffer)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (sameVertices.size() + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// ���� ��ġ�� ���ؽ� ����
		std::vector<unsigned int> firstVertex;
		std::vector<unsigned int> secondVertex;
		firstVertex.resize(sameVertices.size());
		secondVertex.resize(sameVertices.size());
		for (int i = 0; i < sameVertices.size(); i++)
		{
			firstVertex[i] = sameVertices[i].first;
			secondVertex[i] = sameVertices[i].second;
		}
		
		// GPU �޸� �Ҵ� �� CPU �޸𸮸� GPU�� ����
		unsigned int* d_firstVertex;
		unsigned int* d_secondVertex;
		hipMalloc(&d_firstVertex, firstVertex.size() * sizeof(unsigned int));
		hipMalloc(&d_secondVertex, secondVertex.size() * sizeof(unsigned int));
		hipMemcpy(d_firstVertex, firstVertex.data(), firstVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_secondVertex, secondVertex.data(), secondVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		// CUDA �Լ� ����
		processVerticesKernel << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (Vertex*)devPtr, vertexSize);
		cudaStatus = hipDeviceSynchronize(); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}


		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// �޸� ����
		hipFree(d_firstVertex);
		hipFree(d_secondVertex);

		return true;
	}

	bool CudaClothTool::UpdateWorldTransformToID3DBuffer(
		const DirectX::SimpleMath::Matrix& prevWorldTransform,
		const DirectX::SimpleMath::Matrix& nextWorldTrnasform,
		const unsigned int vertexSize,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		DirectX::SimpleMath::Matrix prevTransform = prevWorldTransform.Invert();
		DirectX::SimpleMath::Matrix nextTransform = nextWorldTrnasform;

		SimpleMatrix prevMatrix;
		SimpleMatrix nextMatrix;

		std::memcpy(&prevMatrix, &prevTransform, sizeof(prevMatrix));
		std::memcpy(&nextMatrix, &nextTransform, sizeof(prevMatrix));

		TransformVertices << <blocksPerGrid, threadsPerBlock >> > (particle, prevMatrix, nextMatrix, vertexSize);
		hipError_t cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateWorldTransformToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}
}

