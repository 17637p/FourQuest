#include "hip/hip_runtime.h"
#include "CudaClothTool.h"

#include <hip/hip_runtime.h>
#include <cuda_d3d11_interop.h>
#include <>

#include <iostream>

#include "../FQCommon/FQCommonGraphics.h"

struct SimpleVector3 {
	float x, y, z;
};

struct SimpleMatrix {
	float m[4][4];
};

__device__ physx::PxVec2 Sub(const physx::PxVec2& lhs, const physx::PxVec2& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y };
}

__device__ physx::PxVec4 Sub(const physx::PxVec4& lhs, const physx::PxVec4& rhs) {
	return { lhs.x - rhs.x, lhs.y - rhs.y, lhs.z - rhs.z, lhs.w - rhs.w };
}

__device__ physx::PxVec3 cross(const physx::PxVec4& a, const physx::PxVec4& b) {
	return { a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x };
}

__device__ float DotProduct(const physx::PxVec3& a, const physx::PxVec3& b) {
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ void NormalizeVector(physx::PxVec3& vec) {
	float length = sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
	if (length > 0) {
		vec.x /= length;
		vec.y /= length;
		vec.z /= length;
	}
}

template <typename T>
__device__ SimpleVector3 multiply(SimpleMatrix& mat, const T& vec)
{
	SimpleVector3 result;
	result.x = mat.m[0][0] * vec.x + mat.m[1][0] * vec.y + mat.m[2][0] * vec.z + mat.m[3][0] * 1.0f;
	result.y = mat.m[0][1] * vec.x + mat.m[1][1] * vec.y + mat.m[2][1] * vec.z + mat.m[3][1] * 1.0f;
	result.z = mat.m[0][2] * vec.x + mat.m[1][2] * vec.y + mat.m[2][2] * vec.z + mat.m[3][2] * 1.0f;
	return result;
}

#pragma region UpdateID3D11VertexBuffer
// CUDA Ŀ�� �Լ� ����
template <typename T>
__global__ void UpdateVertex(
	physx::PxVec4* vertices,
	physx::PxVec2* uvs,
	unsigned int vertexSize,
	unsigned int* indices,
	unsigned int indexSize,
	SimpleMatrix invTransform,
	T* buffer)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= indexSize / 3) return;

	unsigned int i0 = indices[idx * 3];
	unsigned int i1 = indices[idx * 3 + 1];
	unsigned int i2 = indices[idx * 3 + 2];

	physx::PxVec4 v0 = vertices[i0];
	physx::PxVec4 v1 = vertices[i1];
	physx::PxVec4 v2 = vertices[i2];

	physx::PxVec2 uv0 = uvs[i0];
	physx::PxVec2 uv1 = uvs[i1];
	physx::PxVec2 uv2 = uvs[i2];

	physx::PxVec4 edge1 = Sub(v1, v0);
	physx::PxVec4 edge2 = Sub(v2, v0);

	physx::PxVec2 deltaUV1 = Sub(uv1, uv0);
	physx::PxVec2 deltaUV2 = Sub(uv2, uv0);

	float f = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);

	physx::PxVec3 tangent;
	tangent.x = f * (deltaUV2.y * edge1.x - deltaUV1.y * edge2.x);
	tangent.y = f * (deltaUV2.y * edge1.y - deltaUV1.y * edge2.y);
	tangent.z = f * (deltaUV2.y * edge1.z - deltaUV1.y * edge2.z);
	NormalizeVector(tangent);

	physx::PxVec3 bitangent;
	bitangent.x = f * (-deltaUV2.x * edge1.x + deltaUV1.x * edge2.x);
	bitangent.y = f * (-deltaUV2.x * edge1.y + deltaUV1.x * edge2.y);
	bitangent.z = f * (-deltaUV2.x * edge1.z + deltaUV1.x * edge2.z);
	NormalizeVector(bitangent);

	physx::PxVec3 normal = cross(edge1, edge2);
	NormalizeVector(normal);

	SimpleVector3 vertex0;
	SimpleVector3 vertex1;
	SimpleVector3 vertex2;
	vertex0.x = v0.x;
	vertex0.y = v0.y;
	vertex0.z = v0.z;
	vertex1.x = v1.x;
	vertex1.y = v1.y;
	vertex1.z = v1.z;
	vertex2.x = v2.x;
	vertex2.y = v2.y;
	vertex2.z = v2.z;
	vertex0 = multiply< SimpleVector3>(invTransform, vertex0);
	vertex1 = multiply< SimpleVector3>(invTransform, vertex1);
	vertex2 = multiply< SimpleVector3>(invTransform, vertex2);

	buffer[i0].Pos.x = vertex0.x;
	buffer[i0].Pos.y = vertex0.y;
	buffer[i0].Pos.z = vertex0.z;
	buffer[i1].Pos.x = vertex1.x;
	buffer[i1].Pos.y = vertex1.y;
	buffer[i1].Pos.z = vertex1.z;
	buffer[i2].Pos.x = vertex2.x;
	buffer[i2].Pos.y = vertex2.y;
	buffer[i2].Pos.z = vertex2.z;

	buffer[i0].Normal.x = normal.x;
	buffer[i0].Normal.y = normal.y;
	buffer[i0].Normal.z = normal.z;
	buffer[i1].Normal.x = normal.x;
	buffer[i1].Normal.y = normal.y;
	buffer[i1].Normal.z = normal.z;
	buffer[i2].Normal.x = normal.x;
	buffer[i2].Normal.y = normal.y;
	buffer[i2].Normal.z = normal.z;

	buffer[i0].Tangent.x = tangent.x;
	buffer[i0].Tangent.y = tangent.y;
	buffer[i0].Tangent.z = tangent.z;
	buffer[i1].Tangent.x = tangent.x;
	buffer[i1].Tangent.y = tangent.y;
	buffer[i1].Tangent.z = tangent.z;
	buffer[i2].Tangent.x = tangent.x;
	buffer[i2].Tangent.y = tangent.y;
	buffer[i2].Tangent.z = tangent.z;
}

template <typename T>
__global__ void UpdateVertexNoIndex(
	physx::PxVec4* vertices,
	unsigned int vertexSize,
	SimpleMatrix invTransform,
	T* buffer)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexSize) return;

	// �ش� ������ �����ɴϴ�
	physx::PxVec4 v = vertices[idx];

	// ��ȯ�� ���� SimpleVector3 ����
	SimpleVector3 vertex;
	vertex.x = v.x;
	vertex.y = v.y;
	vertex.z = v.z;

	// ��ȯ ����� �����Ͽ� ���� ��ġ ������Ʈ
	vertex = multiply<SimpleVector3>(invTransform, vertex);

	// ��ȯ�� ���� ��ġ�� buffer�� ������Ʈ
	buffer[idx].Pos.x = vertex.x;
	buffer[idx].Pos.y = vertex.y;
	buffer[idx].Pos.z = vertex.z;
}

template <typename T>
__global__ void UpdateLerpVertexNoIndex(
	physx::PxVec4* prevVertices,
	physx::PxVec4* currVertices,
	float t,
	unsigned int vertexSize,
	SimpleMatrix invTransform,
	T* buffer)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexSize) return;

	// �ش� ������ �����ɴϴ�
	physx::PxVec4 prevV = prevVertices[idx];
	physx::PxVec4 currV = currVertices[idx];

	// ��ȯ�� ���� SimpleVector3 ���� ( ���� ���� ��ġ, �ֱ� ���� ��ġ ���� ��� )
	SimpleVector3 vertex;
	vertex.x = prevV.x + t * (currV.x - prevV.x);
	vertex.y = prevV.y + t * (currV.y - prevV.y);
	vertex.z = prevV.z + t * (currV.z - prevV.z);

	// ��ȯ ����� �����Ͽ� ���� ��ġ ������Ʈ
	vertex = multiply<SimpleVector3>(invTransform, vertex);

	// ��ȯ�� ���� ��ġ�� buffer�� ������Ʈ
	buffer[idx].Pos.x = vertex.x;
	buffer[idx].Pos.y = vertex.y;
	buffer[idx].Pos.z = vertex.z;
}
#pragma endregion

#pragma region SetID3D11BufferVertexNormal
template <typename T>
__global__ void processVerticesKernel(unsigned int* sameVerticesFirst, unsigned int* sameVerticesSecond, T* buffer, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int first = sameVerticesFirst[idx];
	int second = sameVerticesSecond[idx];

	if (first < size && second < size)
	{
		// �߰����� ����Ͽ� ���� ������ �и�
		float3 normalTemp, tangentTemp, biTangentTemp;

		normalTemp.x = buffer[first].Normal.x + buffer[second].Normal.x;
		normalTemp.y = buffer[first].Normal.y + buffer[second].Normal.y;
		normalTemp.z = buffer[first].Normal.z + buffer[second].Normal.z;

		tangentTemp.x = buffer[first].Tangent.x + buffer[second].Tangent.x;
		tangentTemp.y = buffer[first].Tangent.y + buffer[second].Tangent.y;
		tangentTemp.z = buffer[first].Tangent.z + buffer[second].Tangent.z;

		buffer[first].Normal.x = normalTemp.x;
		buffer[first].Normal.y = normalTemp.y;
		buffer[first].Normal.z = normalTemp.z;
		buffer[second].Normal.x = normalTemp.x;
		buffer[second].Normal.y = normalTemp.y;
		buffer[second].Normal.z = normalTemp.z;

		buffer[first].Tangent.x = tangentTemp.x;
		buffer[first].Tangent.y = tangentTemp.y;
		buffer[first].Tangent.z = tangentTemp.z;
		buffer[second].Tangent.x = tangentTemp.x;
		buffer[second].Tangent.y = tangentTemp.y;
		buffer[second].Tangent.z = tangentTemp.z;
	}
}
#pragma endregion

#pragma region SetClothParticleWorldTransform

// Ŀ�� �Լ�
__global__ void TransformVertices(
	physx::PxVec4* particle,
	SimpleMatrix previousTransformInverse,
	SimpleMatrix newTransform,
	int vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexCount) return;

	SimpleVector3 vertex;
	vertex.x = particle[idx].x;
	vertex.y = particle[idx].y;
	vertex.z = particle[idx].z;

	// ���� worldTransform�� ����� ����
	vertex = multiply< SimpleVector3>(previousTransformInverse, vertex);

	// ���ο� worldTransform ����
	vertex = multiply< SimpleVector3>(newTransform, vertex);

	// ��ȯ�� vertex ����
	particle[idx].x = vertex.x;
	particle[idx].y = vertex.y;
	particle[idx].z = vertex.z;
}
#pragma endregion

#pragma region CopyFromGPUToCPU
template <typename T>
__global__ void CopyVertexDataToCPU(
	physx::PxVec3* vertices,
	physx::PxVec2* uvs,
	SimpleMatrix worldTransform,
	T* ID3D11VertexBuffer,
	unsigned int vertexSize)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= vertexSize) return;

	SimpleVector3 vertex;

	vertex.x = ID3D11VertexBuffer[idx].Pos.x;
	vertex.y = ID3D11VertexBuffer[idx].Pos.y;
	vertex.z = ID3D11VertexBuffer[idx].Pos.z;

	uvs[idx].x = ID3D11VertexBuffer[idx].Tex.x;
	uvs[idx].y = ID3D11VertexBuffer[idx].Tex.y;

	vertex = multiply(worldTransform, vertex);

	vertices[idx].x = vertex.x;
	vertices[idx].y = vertex.y;
	vertices[idx].z = vertex.z;
}
#pragma endregion

#pragma region UpdateSkinnedMeshToPhysXVertex
__global__ void UpdateVertices(Vertex* vertices, physx::PxVec4* particleData, size_t vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vertexCount)
	{
		// �ִϸ��̼� �����͸� ������� ���� ��ǥ ������Ʈ
		particleData[idx].x = vertices[idx].Pos.x;
		particleData[idx].y = vertices[idx].Pos.y;
		particleData[idx].z = vertices[idx].Pos.z;
	}
}
#pragma endregion

#pragma region SetSimulation
__global__ void SetSimulationStopVertex(physx::PxVec4* particleData, size_t vertexCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < vertexCount)
	{
		// �ִϸ��̼� �����͸� ������� ���� ��ǥ ������Ʈ
		particleData[idx].w = 0.f;
	}
}
#pragma endregion

namespace fq::physics
{
	bool CudaClothTool::copyVertexFromGPUToCPU(
		std::vector<DirectX::SimpleMath::Vector3>& vertices,
		std::vector<DirectX::SimpleMath::Vector2>& uvs,
		DirectX::SimpleMath::Matrix& worldTransform,
		hipGraphicsResource* ID3D11VertexBuffer,
		UINT ID3D11VertexStride)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t vertexSize = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &vertexSize, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		// GPU �޸� �Ҵ�
		physx::PxVec3* d_vertices;
		physx::PxVec2* d_uvs;
		hipMalloc(&d_vertices, vertexSize * sizeof(DirectX::SimpleMath::Vector3));
		hipMalloc(&d_uvs, vertexSize * sizeof(DirectX::SimpleMath::Vector2));

		SimpleMatrix Trnasform;
		memcpy(&Trnasform, &worldTransform, sizeof(SimpleMatrix));

		// CUDA �Լ� ����
		if (ID3D11VertexStride == 44)
		{
			CopyVertexDataToCPU <Vertex><< < blocksPerGrid, threadsPerBlock >> > (d_vertices, d_uvs, Trnasform, (Vertex*)devPtr, vertexSize);
		}
		else if (ID3D11VertexStride == 52)
		{
			CopyVertexDataToCPU <Vertex1> << < blocksPerGrid, threadsPerBlock >> > (d_vertices, d_uvs, Trnasform, (Vertex1*)devPtr, vertexSize);
		}
		else if (ID3D11VertexStride == 60)
		{
			CopyVertexDataToCPU <Vertex2> << < blocksPerGrid, threadsPerBlock >> > (d_vertices, d_uvs, Trnasform, (Vertex2*)devPtr, vertexSize);
		}
		
		cudaStatus = hipDeviceSynchronize(); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : "<< cudaStatus << ")" << std::endl;
			return false;
		}

		// ������ ������ ���� CPU �޸𸮷� ����
		hipMemcpy(vertices.data(), d_vertices, vertexSize * sizeof(DirectX::SimpleMath::Vector3), hipMemcpyKind::hipMemcpyDeviceToHost);
		hipMemcpy(uvs.data(), d_uvs, vertexSize * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyDeviceToHost);

		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyVertexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		hipFree(d_vertices);
		hipFree(d_uvs);

		return true;
	}

	bool CudaClothTool::copyIndexFromGPUToCPU(
		std::vector<unsigned int>& indices, 
		hipGraphicsResource* ID3D11IndexBuffer)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11IndexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t indexSize = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &indexSize, ID3D11IndexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		hipMemcpy(indices.data(), devPtr, indexSize * sizeof(unsigned int), hipMemcpyKind::hipMemcpyDeviceToHost);

		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11IndexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}

	bool CudaClothTool::UpdatePhysXDataToID3DBuffer(
		std::vector<DirectX::SimpleMath::Vector3>& vertices, 
		std::vector<unsigned int>& indices, 
		std::vector<DirectX::SimpleMath::Vector2> uvs, 
		DirectX::SimpleMath::Matrix transform,
		hipGraphicsResource* ID3D11VertexBuffer,
		UINT ID3D11VertexStride,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (indices.size() / 3 + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		unsigned int vertexSize = vertices.size();
		unsigned int indexSize = indices.size();

		if (size < sizeof(Vertex) * vertexSize) 
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] Mapped size is smaller than expected!" << std::endl;
			return false;
		}

		// GPU Memory�� �Ҵ��� ����
		physx::PxVec2* d_uvs;
		unsigned int* d_indices;

		// GPU Memory�� �Ҵ� �� ������ ����
		hipMalloc(&d_uvs, uvs.size() * sizeof(DirectX::SimpleMath::Vector2));
		hipMalloc(&d_indices, indices.size() * sizeof(unsigned int));
		hipMemcpy(d_uvs, uvs.data(), uvs.size() * sizeof(DirectX::SimpleMath::Vector2), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_indices, indices.data(), indices.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		SimpleMatrix invTransform;
		std::memcpy(&invTransform, &transform, sizeof(invTransform));

		// CUDA �Լ� ����
		if (ID3D11VertexStride == 44)
		{
			UpdateVertex <Vertex> << <blocksPerGrid, threadsPerBlock >> > (
				particle, d_uvs, vertexSize, d_indices, indexSize, invTransform, (Vertex*)devPtr);
		}
		else if (ID3D11VertexStride == 52)
		{
			UpdateVertex <Vertex1> << <blocksPerGrid, threadsPerBlock >> > (
				particle, d_uvs, vertexSize, d_indices, indexSize, invTransform, (Vertex1*)devPtr);
		}
		else if (ID3D11VertexStride == 60)
		{
			UpdateVertex <Vertex2> << <blocksPerGrid, threadsPerBlock >> > (
				particle, d_uvs, vertexSize, d_indices, indexSize, invTransform, (Vertex2*)devPtr);
		}

		cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// �޸� ����
		hipFree(d_uvs);
		hipFree(d_indices);

		return true;
	}

	bool CudaClothTool::UpdatePhysXDataToID3DVertexBuffer(
		std::vector<DirectX::SimpleMath::Vector3>& vertices, 
		DirectX::SimpleMath::Matrix transform, 
		hipGraphicsResource* ID3D11VertexBuffer,
		UINT ID3D11VertexStride, 
		physx::PxVec4* particle)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		SimpleMatrix invTransform;
		std::memcpy(&invTransform, &transform, sizeof(invTransform));
		unsigned int vertexSize = vertices.size();

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA �Լ� ����
		if (ID3D11VertexStride == 44)
		{
			UpdateVertexNoIndex <Vertex> << <blocksPerGrid, threadsPerBlock >> > (
				particle, vertexSize, invTransform, (Vertex*)devPtr);
		}
		else if (ID3D11VertexStride == 52)
		{
			UpdateVertexNoIndex <Vertex1> << <blocksPerGrid, threadsPerBlock >> > (
				particle, vertexSize, invTransform, (Vertex1*)devPtr);
		}
		else if (ID3D11VertexStride == 60)
		{
			UpdateVertexNoIndex <Vertex2> << <blocksPerGrid, threadsPerBlock >> > (
				particle, vertexSize, invTransform, (Vertex2*)devPtr);
		}

		cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}

	bool CudaClothTool::UpdateNormalToID3DBuffer(
		std::vector<std::pair<unsigned int, unsigned int>>& sameVertices,
		unsigned int vertexSize,
		hipGraphicsResource* ID3D11VertexBuffer,
		UINT ID3D11VertexStride)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (sameVertices.size() + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// ���� ��ġ�� ���ؽ� ����
		std::vector<unsigned int> firstVertex;
		std::vector<unsigned int> secondVertex;
		firstVertex.resize(sameVertices.size());
		secondVertex.resize(sameVertices.size());
		for (int i = 0; i < sameVertices.size(); i++)
		{
			firstVertex[i] = sameVertices[i].first;
			secondVertex[i] = sameVertices[i].second;
		}
		
		// GPU �޸� �Ҵ� �� CPU �޸𸮸� GPU�� ����
		unsigned int* d_firstVertex;
		unsigned int* d_secondVertex;
		hipMalloc(&d_firstVertex, firstVertex.size() * sizeof(unsigned int));
		hipMalloc(&d_secondVertex, secondVertex.size() * sizeof(unsigned int));
		hipMemcpy(d_firstVertex, firstVertex.data(), firstVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_secondVertex, secondVertex.data(), secondVertex.size() * sizeof(unsigned int), hipMemcpyKind::hipMemcpyHostToDevice);

		// CUDA �Լ� ����
		if (ID3D11VertexStride == 44)
		{
			processVerticesKernel <Vertex> << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (Vertex*)devPtr, vertexSize);
		}
		else if (ID3D11VertexStride == 52)
		{
			processVerticesKernel <Vertex1> << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (Vertex1*)devPtr, vertexSize);
		}
		else if (ID3D11VertexStride == 60)
		{
			processVerticesKernel <Vertex2> << <blocksPerGrid, threadsPerBlock >> > (d_firstVertex, d_secondVertex, (Vertex2*)devPtr, vertexSize);
		}

		cudaStatus = hipDeviceSynchronize(); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}


		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer); 
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// �޸� ����
		hipFree(d_firstVertex);
		hipFree(d_secondVertex);

		return true;
	}

	bool CudaClothTool::UpdateWorldTransformToID3DBuffer(
		const DirectX::SimpleMath::Matrix& prevWorldTransform,
		const DirectX::SimpleMath::Matrix& nextWorldTrnasform,
		const unsigned int vertexSize,
		physx::PxVec4* particle)
	{
		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;


		DirectX::SimpleMath::Matrix prevTransform = prevWorldTransform.Invert();
		DirectX::SimpleMath::Matrix nextTransform = nextWorldTrnasform;

		SimpleMatrix prevMatrix;
		SimpleMatrix nextMatrix;

		std::memcpy(&prevMatrix, &prevTransform, sizeof(prevMatrix));
		std::memcpy(&nextMatrix, &nextTransform, sizeof(prevMatrix));

		TransformVertices << <blocksPerGrid, threadsPerBlock >> > (particle, prevMatrix, nextMatrix, vertexSize);
		hipError_t cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateWorldTransformToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}
	bool CudaClothTool::UpdateSkinnedAnimationVertexToPhysicsVertex(
		hipGraphicsResource* ID3D11VertexBuffer, 
		physx::PxVec4* particle, 
		unsigned int vertexSize)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// ��� ����, ������ ���� ����
		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA �Լ� ����
		UpdateVertices << <blocksPerGrid, threadsPerBlock >> > ((Vertex*)devPtr, particle, vertexSize);

		cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}


		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateNormalToID3DBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		return true;
	}

	bool CudaClothTool::UpdateParticleBuffer(const unsigned int vertexSize, DirectX::SimpleMath::Vector4* currParticle, physx::PxVec4* particle)
	{
		// ������ ������ ���� CPU �޸𸮷� ����
		hipError_t cudaStatus = hipMemcpy(currParticle, particle, vertexSize * sizeof(DirectX::SimpleMath::Vector4), hipMemcpyKind::hipMemcpyDeviceToHost);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] UpdateParticleBuffer Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}
		return true;
	}

	bool CudaClothTool::UpdatePhysXDataToID3DVertexBuffer(
		std::vector<DirectX::SimpleMath::Vector4>& prevVertices,
		std::vector<DirectX::SimpleMath::Vector4>& currVertices,
		float deltaTime,
		DirectX::SimpleMath::Matrix invTransform,
		hipGraphicsResource* ID3D11VertexBuffer,
		UINT ID3D11VertexStride)
	{
		// CUDA ���ҽ��� ����
		hipError_t cudaStatus = hipGraphicsMapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		// CUDA ������ ��������
		void* devPtr = nullptr;
		size_t size = 0;
		cudaStatus = hipGraphicsResourceGetMappedPointer(&devPtr, &size, ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error : " << hipGetErrorString(cudaStatus) << ")" << std::endl;
			return false;
		}

		unsigned int vertexSize = currVertices.size();

		if (size < sizeof(Vertex) * vertexSize)
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] Mapped size is smaller than expected!" << std::endl;
			return false;
		}

		SimpleMatrix invMatrix;

		std::memcpy(&invMatrix, &invTransform, sizeof(invTransform));

		// GPU Memory�� �Ҵ��� ����
		physx::PxVec4* d_prevVertices;
		physx::PxVec4* d_currVertices;

		// GPU Memory�� �Ҵ� �� ������ ����
		hipMalloc(&d_prevVertices, vertexSize * sizeof(physx::PxVec4));
		hipMalloc(&d_currVertices, vertexSize * sizeof(physx::PxVec4));
		hipMemcpy(d_prevVertices, prevVertices.data(), prevVertices.size() * sizeof(physx::PxVec4), hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(d_currVertices, currVertices.data(), currVertices.size() * sizeof(physx::PxVec4), hipMemcpyKind::hipMemcpyHostToDevice);

		int threadsPerBlock = 256;
		int blocksPerGrid = (vertexSize + threadsPerBlock - 1) / threadsPerBlock;

		// CUDA �Լ� ����
		if (ID3D11VertexStride == 44)
		{
			UpdateLerpVertexNoIndex <Vertex> << <blocksPerGrid, threadsPerBlock >> > (
				d_prevVertices, d_currVertices, deltaTime, vertexSize, invMatrix, (Vertex*)devPtr);
		}
		else if (ID3D11VertexStride == 52)
		{
			UpdateLerpVertexNoIndex <Vertex1> << <blocksPerGrid, threadsPerBlock >> > (
				d_prevVertices, d_currVertices, deltaTime, vertexSize, invMatrix, (Vertex1*)devPtr);
		}
		else if (ID3D11VertexStride == 60)
		{
			UpdateLerpVertexNoIndex <Vertex2> << <blocksPerGrid, threadsPerBlock >> > (
				d_prevVertices, d_currVertices, deltaTime, vertexSize, invMatrix, (Vertex2*)devPtr);
		}

		cudaStatus = hipDeviceSynchronize();
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// CUDA ���ҽ��� �����
		cudaStatus = hipGraphicsUnmapResources(1, &ID3D11VertexBuffer);
		if (!(cudaStatus == hipSuccess))
		{
			std::cerr << "[CudaClothTool(" << __LINE__ << ")] copyIndexFromGPUToCPU Error(Error Code : " << cudaStatus << ")" << std::endl;
			return false;
		}

		// �޸� ����
		hipFree(d_prevVertices);
		hipFree(d_currVertices);

		return true;
	}
}

